#include "hip/hip_runtime.h"


#include "tfcc_cudaconvolutioninterface.h"

#include "exceptions/tfcc_cudaruntimeerror.h"
#include "exceptions/tfcc_cudnnruntimeerror.h"
#include "exceptions/tfcc_invalidargumenterror.h"
#include "exceptions/tfcc_notimplementederror.h"
#include "framework/tfcc_cudadevice.h"
#include "framework/tfcc_cudasession.h"
#include "framework/tfcc_cudatypes.h"
#include "framework/tfcc_session.h"
#include "framework/tfcc_types.h"
#include "utils/tfcc_cudnnutils.h"

namespace tfcc {

/**
 * [s1, s2, s3] => [s1, s3, s2]
 */
template <class T>
static __global__ void _cuda_convolution_transpose(const T* a, unsigned s1, unsigned s2, unsigned s3, T* b) {
  const unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
  const unsigned skip = blockDim.x * gridDim.x;
  const unsigned total = s1 * s2 * s3;

  for (unsigned i = tid; i < total; i += skip) {
    unsigned ns1 = (i / (s2 * s3)) % s1;
    unsigned ns2 = (i / s3) % s2;
    unsigned ns3 = i % s3;

    unsigned pos = ns1 * s2 * s3 + ns3 * s2 + ns2;
    b[pos] = a[i];
  }
}

template <class T>
CUDAConvolutionInterface<T>::CUDAConvolutionInterface(const CUDADeviceProperty& property)
  : _property(property) {
}

template <class T>
CUDAConvolutionInterface<T>::~CUDAConvolutionInterface() {
}

template <class T>
Variable<T> CUDAConvolutionInterface<T>::conv2d(
    const Tensor<T>& input, bool nhwc,
    const Tensor<T>& kernel,
    unsigned paddingHeight, unsigned paddingWidth,
    unsigned strideHeight, unsigned strideWidth,
    unsigned dilateHeight, unsigned dilateWidth) {
  unsigned batch = input.shape(0);
  unsigned outChannels = kernel.shape(0);
  unsigned inChannels = kernel.shape(1);
  unsigned inHeight = input.shape(nhwc ? 1 : 2);
  unsigned inWidth = input.shape(nhwc ? 2 : 3);
  unsigned kernelHeight = kernel.shape(2);
  unsigned kernelWidth = kernel.shape(3);
  unsigned outHeight = (inHeight - kernelHeight + 2 * paddingHeight) / strideHeight + 1;
  unsigned outWidth = (inWidth - kernelWidth + 2 * paddingWidth) / strideWidth + 1;
  hipdnnDataType_t dataType = CUDATypeTraits<T>::getCUDNNType();

  tfcc::CUDASession* session = static_cast<tfcc::CUDASession*>(Session::getThreadDefault());

  Variable<T> output(nhwc ? Shape({batch, outHeight, outWidth, outChannels}) : Shape({batch, outChannels, outHeight, outWidth}));

  hipdnnTensorDescriptor_t inputDescriptor;
  hipdnnStatus_t ret = hipdnnCreateTensorDescriptor(&inputDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  CudnnTensorDescriptorGuard inputGuard(&inputDescriptor);

  ret = hipdnnSetTensor4dDescriptor(
      inputDescriptor,
      nhwc ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      dataType,
      batch,
      inChannels,
      inHeight,
      inWidth);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  // output tensor
  hipdnnTensorDescriptor_t outputDescriptor;
  ret = hipdnnCreateTensorDescriptor(&outputDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  CudnnTensorDescriptorGuard outputGuard(&outputDescriptor);

  ret = hipdnnSetTensor4dDescriptor(
      outputDescriptor,
      nhwc ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      dataType,
      batch,
      outChannels,
      outHeight,
      outWidth);

  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  // kernel tensor
  hipdnnFilterDescriptor_t kernelDescriptor;
  ret = hipdnnCreateFilterDescriptor(&kernelDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  CudnnFilterDescriptorGuard kernelGuard(&kernelDescriptor);

  ret = hipdnnSetFilter4dDescriptor(
      kernelDescriptor,
      dataType,
      HIPDNN_TENSOR_NCHW,
      outChannels,
      inChannels,
      kernelHeight,
      kernelWidth);

  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  // conv descriptor
  hipdnnConvolutionDescriptor_t convolutionDescriptor;
  ret = hipdnnCreateConvolutionDescriptor(&convolutionDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  CudnnConvolutionDescriptorGuard convGuard(&convolutionDescriptor);

  tfcc::CUDADevice* device = static_cast<tfcc::CUDADevice*>(tfcc::Device::getThreadDefault());
#ifdef TFCC_USE_TENSOR_CORE
  if (device->isTensorCoreEnabled()) {
    ret = hipdnnSetConvolutionMathType(convolutionDescriptor, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION);  //HIPDNN_TENSOR_OP_MATH);
    if (ret != HIPDNN_STATUS_SUCCESS)
      throw CUDNNRuntimeError(ret);
  }
#endif

  ret = hipdnnSetConvolution2dDescriptor(
      convolutionDescriptor,
      paddingHeight,
      paddingWidth,
      strideHeight,
      strideWidth,
      dilateHeight,
      dilateWidth,
      HIPDNN_CROSS_CORRELATION,
      dataType);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  // conv algorithm
  hipdnnConvolutionFwdAlgo_t convolutionAlgorithm;

  if (device->isTensorCoreEnabled()) {
    convolutionAlgorithm = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
  } else {
    ret = hipdnnGetConvolutionForwardAlgorithm(
        session->getImpl()->cudnnHandle(),
        inputDescriptor,
        kernelDescriptor,
        convolutionDescriptor,
        outputDescriptor,
        HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE,
        0,
        &convolutionAlgorithm);
    if (ret != HIPDNN_STATUS_SUCCESS)
      throw CUDNNRuntimeError(ret);
  }
  // alloc workspace memory
  size_t workspaceBytes = 0;
  ret = hipdnnGetConvolutionForwardWorkspaceSize(
      session->getImpl()->cudnnHandle(),
      inputDescriptor,
      kernelDescriptor,
      convolutionDescriptor,
      outputDescriptor,
      convolutionAlgorithm,
      &workspaceBytes);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  unsigned tmpSize = static_cast<unsigned>((workspaceBytes + sizeof(T) - 1) / sizeof(T));
  tmpSize = tmpSize == 0 ? 1 : tmpSize;
  Variable<T> tmp({
      tmpSize,
  });

  // run
  T alpha = static_cast<T>(1.0), beta = static_cast<T>(0.0);
  ret = hipdnnConvolutionForward(
      session->getImpl()->cudnnHandle(),
      &alpha,
      inputDescriptor,
      input.data(),
      kernelDescriptor,
      kernel.data(),
      convolutionDescriptor,
      convolutionAlgorithm,
      tmp.data(),
      workspaceBytes,
      &beta,
      outputDescriptor,
      output.data());
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  return output;
}

template <class T>
Variable<T> CUDAConvolutionInterface<T>::conv2d(
    const Tensor<T>& input, bool nhwc,
    const Tensor<T>& kernel,
    unsigned paddingHeight, unsigned paddingWidth,
    unsigned strideHeight, unsigned strideWidth) {
  unsigned batch = input.shape(0);
  unsigned outChannels = kernel.shape(0);
  unsigned inChannels = kernel.shape(1);
  unsigned inHeight = input.shape(nhwc ? 1 : 2);
  unsigned inWidth = input.shape(nhwc ? 2 : 3);
  unsigned kernelHeight = kernel.shape(2);
  unsigned kernelWidth = kernel.shape(3);
  unsigned outHeight = (inHeight - kernelHeight + 2 * paddingHeight) / strideHeight + 1;
  unsigned outWidth = (inWidth - kernelWidth + 2 * paddingWidth) / strideWidth + 1;
  hipdnnDataType_t dataType = CUDATypeTraits<T>::getCUDNNType();

  tfcc::CUDASession* session = static_cast<tfcc::CUDASession*>(Session::getThreadDefault());

  Variable<T> output(nhwc ? Shape({batch, outHeight, outWidth, outChannels}) : Shape({batch, outChannels, outHeight, outWidth}));

  hipdnnTensorDescriptor_t inputDescriptor;
  hipdnnStatus_t ret = hipdnnCreateTensorDescriptor(&inputDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  CudnnTensorDescriptorGuard inputGuard(&inputDescriptor);

  ret = hipdnnSetTensor4dDescriptor(
      inputDescriptor,
      nhwc ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      dataType,
      batch,
      inChannels,
      inHeight,
      inWidth);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  // output tensor
  hipdnnTensorDescriptor_t outputDescriptor;
  ret = hipdnnCreateTensorDescriptor(&outputDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  CudnnTensorDescriptorGuard outputGuard(&outputDescriptor);

  ret = hipdnnSetTensor4dDescriptor(
      outputDescriptor,
      nhwc ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      dataType,
      batch,
      outChannels,
      outHeight,
      outWidth);

  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  // kernel tensor
  hipdnnFilterDescriptor_t kernelDescriptor;
  ret = hipdnnCreateFilterDescriptor(&kernelDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  CudnnFilterDescriptorGuard kernelGuard(&kernelDescriptor);

  ret = hipdnnSetFilter4dDescriptor(
      kernelDescriptor,
      dataType,
      HIPDNN_TENSOR_NCHW,
      outChannels,
      inChannels,
      kernelHeight,
      kernelWidth);

  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  // conv descriptor
  hipdnnConvolutionDescriptor_t convolutionDescriptor;
  ret = hipdnnCreateConvolutionDescriptor(&convolutionDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  CudnnConvolutionDescriptorGuard convGuard(&convolutionDescriptor);

  tfcc::CUDADevice* device = static_cast<tfcc::CUDADevice*>(tfcc::Device::getThreadDefault());
#ifdef TFCC_USE_TENSOR_CORE
  if (device->isTensorCoreEnabled()) {
    ret = hipdnnSetConvolutionMathType(convolutionDescriptor, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION);  //HIPDNN_TENSOR_OP_MATH);
    if (ret != HIPDNN_STATUS_SUCCESS)
      throw CUDNNRuntimeError(ret);
  }
#endif

  ret = hipdnnSetConvolution2dDescriptor(
      convolutionDescriptor,
      paddingHeight,
      paddingWidth,
      strideHeight,
      strideWidth,
      1,
      1,
      HIPDNN_CROSS_CORRELATION,
      dataType);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  // conv algorithm
  hipdnnConvolutionFwdAlgo_t convolutionAlgorithm;

  if (device->isTensorCoreEnabled()) {
    convolutionAlgorithm = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
  } else {
    ret = hipdnnGetConvolutionForwardAlgorithm(
        session->getImpl()->cudnnHandle(),
        inputDescriptor,
        kernelDescriptor,
        convolutionDescriptor,
        outputDescriptor,
        HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE,
        0,
        &convolutionAlgorithm);
    if (ret != HIPDNN_STATUS_SUCCESS)
      throw CUDNNRuntimeError(ret);
  }
  // alloc workspace memory
  size_t workspaceBytes = 0;
  ret = hipdnnGetConvolutionForwardWorkspaceSize(
      session->getImpl()->cudnnHandle(),
      inputDescriptor,
      kernelDescriptor,
      convolutionDescriptor,
      outputDescriptor,
      convolutionAlgorithm,
      &workspaceBytes);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  unsigned tmpSize = static_cast<unsigned>((workspaceBytes + sizeof(T) - 1) / sizeof(T));
  tmpSize = tmpSize == 0 ? 1 : tmpSize;
  Variable<T> tmp({
      tmpSize,
  });

  // run
  T alpha = static_cast<T>(1.0), beta = static_cast<T>(0.0);
  ret = hipdnnConvolutionForward(
      session->getImpl()->cudnnHandle(),
      &alpha,
      inputDescriptor,
      input.data(),
      kernelDescriptor,
      kernel.data(),
      convolutionDescriptor,
      convolutionAlgorithm,
      tmp.data(),
      workspaceBytes,
      &beta,
      outputDescriptor,
      output.data());
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  return output;
}

template <class T>
Variable<T> CUDAConvolutionInterface<T>::conv2dBackwardData(
    const Tensor<T>& input, bool nhwc,
    const Tensor<T>& kernel,
    unsigned paddingHeight, unsigned paddingWidth,
    unsigned strideHeight, unsigned strideWidth) {
  if (!nhwc) {
    return conv2dBackwardDataNCHW(input, kernel, paddingHeight, paddingWidth, strideHeight, strideWidth);
  }
  Variable<T> realInput = nhwc2nchw(input);
  Variable<T> output = conv2dBackwardDataNCHW(realInput, kernel, paddingHeight, paddingWidth, strideHeight, strideWidth);
  return nchw2nhwc(output);
}

template <class T>
Variable<T> CUDAConvolutionInterface<T>::maxPool2d(
    const Tensor<T>& input, bool nhwc,
    unsigned kernelHeight, unsigned kernelWidth,
    unsigned paddingHeight, unsigned paddingWidth,
    unsigned strideHeight, unsigned strideWidth) {
  unsigned batch = input.shape(0);
  unsigned outChannels = nhwc ? input.shape(3) : input.shape(1);
  unsigned inChannels = outChannels;
  unsigned inHeight = nhwc ? input.shape(1) : input.shape(2);
  unsigned inWidth = nhwc ? input.shape(2) : input.shape(3);
  unsigned outHeight = (inHeight - kernelHeight + 2 * paddingHeight) / strideHeight + 1;
  unsigned outWidth = (inWidth - kernelWidth + 2 * paddingWidth) / strideWidth + 1;
  hipdnnDataType_t dataType = CUDATypeTraits<T>::getCUDNNType();

  tfcc::CUDASession* session = static_cast<tfcc::CUDASession*>(Session::getThreadDefault());

  Variable<T> output(nhwc ? Shape({batch, outHeight, outWidth, outChannels}) : Shape({batch, outChannels, outHeight, outWidth}));

  hipdnnTensorDescriptor_t inputDescriptor;
  hipdnnStatus_t ret = hipdnnCreateTensorDescriptor(&inputDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  ret = hipdnnSetTensor4dDescriptor(
      inputDescriptor,
      nhwc ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      dataType,
      batch,
      inChannels,
      inHeight,
      inWidth);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  hipdnnTensorDescriptor_t outputDescriptor;
  ret = hipdnnCreateTensorDescriptor(&outputDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  ret = hipdnnSetTensor4dDescriptor(
      outputDescriptor,
      nhwc ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      dataType,
      batch,
      outChannels,
      outHeight,
      outWidth);

  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  hipdnnPoolingDescriptor_t poolingDescriptor;
  ret = hipdnnCreatePoolingDescriptor(&poolingDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  ret = hipdnnSetPooling2dDescriptor(
      poolingDescriptor,
      HIPDNN_POOLING_MAX,
      HIPDNN_NOT_PROPAGATE_NAN,
      kernelHeight,
      kernelWidth,
      paddingHeight,
      paddingWidth,
      strideHeight,
      strideWidth);

  T alpha = static_cast<T>(1.0), beta = static_cast<T>(0.0);
  ret = hipdnnPoolingForward(
      session->getImpl()->cudnnHandle(),
      poolingDescriptor,
      &alpha,
      inputDescriptor,
      input.data(),
      &beta,
      outputDescriptor,
      output.data());
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  hipdnnDestroyTensorDescriptor(inputDescriptor);
  hipdnnDestroyTensorDescriptor(outputDescriptor);
  hipdnnDestroyPoolingDescriptor(poolingDescriptor);
  return output;
}

template <class T>
Variable<T> CUDAConvolutionInterface<T>::avgPool2d(
    const Tensor<T>& input, bool nhwc,
    unsigned kernelHeight, unsigned kernelWidth,
    unsigned paddingHeight, unsigned paddingWidth,
    unsigned strideHeight, unsigned strideWidth) {
  unsigned batch = input.shape(0);
  unsigned outChannels = nhwc ? input.shape(3) : input.shape(1);
  unsigned inChannels = outChannels;
  unsigned inHeight = nhwc ? input.shape(1) : input.shape(2);
  unsigned inWidth = nhwc ? input.shape(2) : input.shape(3);
  unsigned outHeight = (inHeight - kernelHeight + 2 * paddingHeight) / strideHeight + 1;
  unsigned outWidth = (inWidth - kernelWidth + 2 * paddingWidth) / strideWidth + 1;
  hipdnnDataType_t dataType = CUDATypeTraits<T>::getCUDNNType();

  tfcc::CUDASession* session = static_cast<tfcc::CUDASession*>(Session::getThreadDefault());

  Variable<T> output(nhwc ? Shape({batch, outHeight, outWidth, outChannels}) : Shape({batch, outChannels, outHeight, outWidth}));

  hipdnnTensorDescriptor_t inputDescriptor;
  hipdnnStatus_t ret = hipdnnCreateTensorDescriptor(&inputDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  ret = hipdnnSetTensor4dDescriptor(
      inputDescriptor,
      nhwc ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      dataType,
      batch,
      inChannels,
      inHeight,
      inWidth);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  hipdnnTensorDescriptor_t outputDescriptor;
  ret = hipdnnCreateTensorDescriptor(&outputDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  ret = hipdnnSetTensor4dDescriptor(
      outputDescriptor,
      nhwc ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      dataType,
      batch,
      outChannels,
      outHeight,
      outWidth);

  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  hipdnnPoolingDescriptor_t poolingDescriptor;
  ret = hipdnnCreatePoolingDescriptor(&poolingDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  ret = hipdnnSetPooling2dDescriptor(
      poolingDescriptor,
      HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
      HIPDNN_NOT_PROPAGATE_NAN,
      kernelHeight,
      kernelWidth,
      paddingHeight,
      paddingWidth,
      strideHeight,
      strideWidth);

  T alpha = static_cast<T>(1.0), beta = static_cast<T>(0.0);
  ret = hipdnnPoolingForward(
      session->getImpl()->cudnnHandle(),
      poolingDescriptor,
      &alpha,
      inputDescriptor,
      input.data(),
      &beta,
      outputDescriptor,
      output.data());
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  hipdnnDestroyTensorDescriptor(inputDescriptor);
  hipdnnDestroyTensorDescriptor(outputDescriptor);
  hipdnnDestroyPoolingDescriptor(poolingDescriptor);
  return output;
}

template <class T>
Variable<T> CUDAConvolutionInterface<T>::conv2dBackwardDataNCHW(
    const Tensor<T>& input,
    const Tensor<T>& kernel,
    unsigned paddingHeight, unsigned paddingWidth,
    unsigned strideHeight, unsigned strideWidth) {
  unsigned batch = input.shape(0);
  unsigned inHeight = input.shape(2);
  unsigned inWidth = input.shape(3);
  unsigned kernelHeight = kernel.shape(2);
  unsigned kernelWidth = kernel.shape(3);
  unsigned outHeight = (inHeight - 1) * strideHeight + kernelHeight - 2 * paddingHeight;
  unsigned outWidth = (inWidth - 1) * strideWidth + kernelWidth - 2 * paddingWidth;
  unsigned inChannels = kernel.shape(0);
  unsigned outChannels = kernel.shape(1);
  hipdnnDataType_t dataType = CUDATypeTraits<T>::getCUDNNType();

  tfcc::CUDASession* session = static_cast<tfcc::CUDASession*>(Session::getThreadDefault());
  Variable<T> output({batch, outChannels, outHeight, outWidth});

  hipdnnTensorDescriptor_t outputDescriptor;
  hipdnnStatus_t ret = hipdnnCreateTensorDescriptor(&outputDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  ret = hipdnnSetTensor4dDescriptor(
      outputDescriptor,
      HIPDNN_TENSOR_NCHW,
      dataType,
      batch,
      outChannels,
      outHeight,
      outWidth);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  hipdnnTensorDescriptor_t inputDescriptor;
  ret = hipdnnCreateTensorDescriptor(&inputDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  ret = hipdnnSetTensor4dDescriptor(
      inputDescriptor,
      HIPDNN_TENSOR_NCHW,
      dataType,
      batch,
      inChannels,
      inHeight,
      inWidth);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  hipdnnFilterDescriptor_t kernelDescriptor;
  ret = hipdnnCreateFilterDescriptor(&kernelDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  ret = hipdnnSetFilter4dDescriptor(
      kernelDescriptor,
      dataType,
      HIPDNN_TENSOR_NCHW,
      inChannels,
      outChannels,
      kernelHeight,
      kernelWidth);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  hipdnnConvolutionDescriptor_t convolutionDescriptor;
  ret = hipdnnCreateConvolutionDescriptor(&convolutionDescriptor);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  ret = hipdnnSetConvolution2dDescriptor(
      convolutionDescriptor,
      paddingHeight,
      paddingWidth,
      strideHeight,
      strideWidth,
      1,
      1,
      HIPDNN_CROSS_CORRELATION,
      dataType);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  hipdnnConvolutionBwdDataAlgo_t convolutionAlgorithm;
  ret = hipdnnGetConvolutionBackwardDataAlgorithm(
      session->getImpl()->cudnnHandle(),
      kernelDescriptor,
      inputDescriptor,
      convolutionDescriptor,
      outputDescriptor,
      HIPDNN_CONVOLUTION_BWD_DATA_NO_WORKSPACE,
      0,
      &convolutionAlgorithm);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  size_t workspaceBytes = 0;
  ret = hipdnnGetConvolutionBackwardDataWorkspaceSize(
      session->getImpl()->cudnnHandle(),
      kernelDescriptor,
      inputDescriptor,
      convolutionDescriptor,
      outputDescriptor,
      convolutionAlgorithm,
      &workspaceBytes);
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);
  unsigned tmpSize = static_cast<unsigned>((workspaceBytes + sizeof(T) - 1) / sizeof(T));
  tmpSize = tmpSize == 0 ? 1 : tmpSize;
  Variable<T> tmp({
      tmpSize,
  });

  T alpha = static_cast<T>(1.0), beta = static_cast<T>(0.0);
  ret = hipdnnConvolutionBackwardData(
      session->getImpl()->cudnnHandle(),
      &alpha,
      kernelDescriptor,
      kernel.data(),
      inputDescriptor,
      input.data(),
      convolutionDescriptor,
      convolutionAlgorithm,
      tmp.data(),
      workspaceBytes,
      &beta,
      outputDescriptor,
      output.data());
  if (ret != HIPDNN_STATUS_SUCCESS)
    throw CUDNNRuntimeError(ret);

  hipdnnDestroyTensorDescriptor(inputDescriptor);
  hipdnnDestroyTensorDescriptor(outputDescriptor);
  hipdnnDestroyFilterDescriptor(kernelDescriptor);
  hipdnnDestroyConvolutionDescriptor(convolutionDescriptor);

  return output;
}

template <class T>
Variable<T> CUDAConvolutionInterface<T>::nhwc2nchw(const Tensor<T>& a) {
  Variable<T> result({a.shape(0), a.shape(3), a.shape(1), a.shape(2)});
  size_t blockCount, threadCount;
  std::tie(blockCount, threadCount) = _property.getSuitableKernelSize(result.size());

  tfcc::CUDASession* session = static_cast<tfcc::CUDASession*>(Session::getThreadDefault());
  _cuda_convolution_transpose<<<blockCount, threadCount, 0, session->getImpl()->cudaStream()>>>(
      a.data(),
      a.shape(0), a.shape(1) * a.shape(2), a.shape(3),
      result.data());
  hipError_t ret = hipGetLastError();
  if (ret != hipSuccess)
    throw CUDARuntimeError(ret);

  return result;
}

template <class T>
Variable<T> CUDAConvolutionInterface<T>::nchw2nhwc(const Tensor<T>& a) {
  Variable<T> result({a.shape(0), a.shape(2), a.shape(3), a.shape(1)});
  size_t blockCount, threadCount;
  std::tie(blockCount, threadCount) = _property.getSuitableKernelSize(result.size());

  tfcc::CUDASession* session = static_cast<tfcc::CUDASession*>(Session::getThreadDefault());
  _cuda_convolution_transpose<<<blockCount, threadCount, 0, session->getImpl()->cudaStream()>>>(
      a.data(),
      a.shape(0), a.shape(1), a.shape(2) * a.shape(3),
      result.data());
  hipError_t ret = hipGetLastError();
  if (ret != hipSuccess)
    throw CUDARuntimeError(ret);

  return result;
}

#define DEFINE_FUNC(type) template class CUDAConvolutionInterface<type>;

TFCC_FOR_ALL_TYPES(DEFINE_FUNC);

}  // namespace tfcc
